
#include <hip/hip_runtime.h>
#include <chrono>
#include <thread>
#include <iostream>
#include <ncurses.h>

__attribute__((noinline))
void _abortError(const char* msg, const char* fname, int line)
{
    hipError_t err = hipGetLastError();
    std::clog << fname << ": " << "line: " << line << ": " << msg << '\n';
    std::clog << "Error " << hipGetErrorName(err) << ": "
              << hipGetErrorString(err) << '\n';
    std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)

__global__
void compute_iteration(char* buffer, char* out_buffer, size_t pitch,
                       size_t pitch_out, int width, int height)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    int left_x = (x - 1 + width) % width;
    int right_x = (x + 1) % width;
    int up_y = (y - 1 + height) % height;
    int down_y = (y + 1) % height;
    char n_alive = buffer[up_y * pitch + left_x] + buffer[up_y * pitch + x] +
        buffer[up_y * pitch + right_x] + buffer[y * pitch + left_x] +
        buffer[y * pitch + right_x] + buffer[down_y * pitch + left_x] +
        buffer[down_y * pitch + x] + buffer[down_y * pitch + right_x];

    out_buffer[y * pitch + x] = n_alive == 3 || (buffer[y * pitch + x]
                                                 && n_alive == 2);
}

void display(char *dev_buffer, size_t pitch, int width, int height)
{
    auto map = new char[width * height];
    size_t pitch2 = width * sizeof(char);

    hipMemcpy2D(map, pitch2, dev_buffer, pitch, width, height,
               hipMemcpyDeviceToHost);

    wmove(stdscr, 0, 0);
    for (size_t y = 0; y < height; ++y)
    {
        for (size_t x = 0; x < width; ++x)
        {
            if (map[y * width + x])
                waddch(stdscr, '.');
            else
                waddch(stdscr, 'x');
        }
        waddch(stdscr, '\n');
    }
    wrefresh(stdscr);

    delete map;
}

void run_compute_iteration(char* dev_buffer, char* out_dev_buffer,
                           size_t pitch, size_t pitch_out, int width,
                           int height, int n_iterations = 1000)
{
    constexpr int block_size = 32;
    int w = std::ceil(1.f * width / block_size);
    int h = std::ceil(1.f * height / block_size);

    dim3 dimGrid(w, h);
    dim3 dimBlock(block_size, block_size);

    auto map = new char[width * height];
    memset(map, 0, width * height);
    size_t pitch2 = width * sizeof(char);
    // Glider hardcoded
    map[1 * height + 2] = 1;
    map[2 * height + 3] = 1;
    map[3 * height + 1] = 1;
    map[3 * height + 2] = 1;
    map[3 * height + 3] = 1;

    hipMemcpy2D(dev_buffer, pitch, map, pitch2, width * sizeof(char), height,
                 hipMemcpyHostToDevice);
    delete map;


    for (int i = 0; i < n_iterations; ++i)
    {
        compute_iteration<<<dimGrid, dimBlock>>>(
                dev_buffer, out_dev_buffer, pitch, pitch_out, width, height);
        std::swap(dev_buffer, out_dev_buffer);
        display(dev_buffer, pitch, width, height);
        std::this_thread::sleep_for(std::chrono::milliseconds(1000));
    }

    if (hipPeekAtLastError())
        abortError("Computation error");
}

int main(int argc, char *argv[])
{
    // FIXME: Parse Game of life plaintext format file

    //constexpr int width = 1024;
    //constexpr int height = 768;
    constexpr int width = 10;
    constexpr int height = 10;

    hipError_t rc = hipSuccess;

    // Allocate device memory
    char* dev_buffer;
    char* out_dev_buffer;
    size_t pitch;
    size_t pitch_out;

    rc = hipMallocPitch(&dev_buffer, &pitch, width * sizeof(char), height);
    if (rc)
        abortError("Fail buffer allocation");

    rc = hipMemset2D(dev_buffer, pitch, 0, width, height);
    if (rc)
        abortError("Fail buffer memset");

    rc = hipMallocPitch(&out_dev_buffer, &pitch_out, width * sizeof(char),
                         height);
    if (rc)
        abortError("Fail output buffer allocation");

    initscr();

    run_compute_iteration(dev_buffer, out_dev_buffer, pitch, pitch_out, width,
                          height);

    rc = hipFree(dev_buffer);
    if (rc)
        abortError("Unable to free buffer");

    rc = hipFree(out_dev_buffer);
    if (rc)
        abortError("Unable to free output buffer");

    endwin();

    return 0;
}
